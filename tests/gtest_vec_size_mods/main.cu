#include <iostream>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <gtest/gtest.h>
#include "../../src/splitvector/splitvec.h"
#include <hip/hip_runtime_api.h>

typedef split::SplitVector<int> vec ;
typedef split::SplitVector<split::SplitVector<int>> vec2d ;


uint32_t fnv_1a(const void* chunk, size_t bytes){
    assert(chunk);
    uint32_t h = 2166136261ul;
    const unsigned char* ptr = (const unsigned char*)chunk;
    while (bytes--){
       h = (h ^ *ptr++) * 16777619ul;
    }
    return h ;
 }

TEST(Ctor,Vec_of_Vec){
   vec A(100);
   EXPECT_TRUE(A.size()==100);
   EXPECT_TRUE(A.capacity()==100);
   for (size_t i=0; i<100;i++){
      A.push_back(2);
   }
   EXPECT_TRUE(A.size()==200);
   EXPECT_TRUE(A.capacity()>200);
}

TEST(CtorStdVec,stdvec){


   std::vector<int> stdvec{1,2,3,4,5,6};
   vec splitvec{1,2,3,4,5,6};
   auto hash0=fnv_1a(stdvec.data(),stdvec.size()*sizeof(int));
   auto hash1=fnv_1a(splitvec.data(),splitvec.size()*sizeof(int));
   EXPECT_TRUE(hash0==hash1);

   vec splitvec_2(stdvec);
   auto hash2=fnv_1a(splitvec_2.data(),splitvec_2.size()*sizeof(int));
   EXPECT_TRUE(hash0==hash1&& hash1==hash2);
   EXPECT_TRUE(splitvec_2.size()==stdvec.size());
   EXPECT_FALSE(splitvec_2.data()==stdvec.data());

   
   for (size_t i =0 ; i<100;i++){
      stdvec.push_back(i);
      splitvec.push_back(i);
      splitvec_2.push_back(i);
   }


   stdvec.pop_back();
   splitvec.pop_back();
   splitvec_2.pop_back();

   auto hash3=fnv_1a(stdvec.data(),stdvec.size()*sizeof(int));
   auto hash4=fnv_1a(splitvec.data(),splitvec.size()*sizeof(int));
   auto hash5=fnv_1a(splitvec_2.data(),splitvec_2.size()*sizeof(int));
   EXPECT_TRUE(hash3==hash4 && hash4==hash5);
}




int main(int argc, char* argv[]){
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}
